#include "hip/hip_runtime.h"
/*
*	In His Exalted Name
*	Vector Addition - Sequential Code
*	Ahmad Siavashi, Email: siavashi@aut.ac.ir
*	21/05/2018
*/
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include ""
#include <math.h>

#pragma comment(lib, "winmm.lib")
void fillVector(int * v, size_t n);
void addVector(int * a, int *b, int *c, size_t n);
void printVector(int * v, size_t n);
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
int *dev_a = 0;
int *dev_b = 0;
int *dev_c = 0;
hipError_t cudaStatus;
__device__ const unsigned int vectorSize = 1023 * 513;

int main()
{

	const bool isCudaMode = true;

	int *a = new int[vectorSize];
	int *b = new int[vectorSize];
	int *c = new int[vectorSize];


	fillVector(a, vectorSize);
	fillVector(b, vectorSize);

	hipEvent_t start, stop;
	float elapsed_time;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	if (isCudaMode) {
		addWithCuda(c, a, b, vectorSize);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
	}
	else
	{
		addVector(a, b, c, vectorSize);
	}

	//end_time = clock
	hipEventElapsedTime(&elapsed_time, start, stop);
	printf("Elapsed time is %lf\n", elapsed_time);
	//printVector(c, vectorSize);
	system("pause");
	return EXIT_SUCCESS;
}

// Fills a vector with data
void fillVector(int * v, size_t n) {
	int i;
	for (i = 0; i < n; i++) {
		v[i] = i;
	}
}

// Adds two vectors
void addVector(int * a, int *b, int *c, size_t n) {
	int i;

	for (i = 0; i < n; i++) {
		c[i] = a[i] + b[i];
	}
}

// Prints a vector to the stdout.
void printVector(int * v, size_t n) {
	int i;
	
	printf("[-] Vector elements: ");
	for (i = 0; i < n; i++) {
		printf("%d, ", v[i]);
	}
	printf("\b\b  \n");
}

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int g_tId = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int warpId = threadIdx.x / warpSize;

	if ( g_tId < vectorSize) {
		c[g_tId] = a[g_tId] + b[g_tId];
		printf("thread id %d , warp Id %d , block id %d\n", g_tId, warpId,blockIdx.x);
	}
}

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size) {
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
	}
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return cudaStatus;
	}
	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return cudaStatus;
	}
	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
		return cudaStatus;
	}
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return cudaStatus;
	}
	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return cudaStatus;
	}
	addKernel << <  ceil(size / 1024), 1024 >> >(dev_c, dev_a, dev_b);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return cudaStatus;
	}
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
		return cudaStatus;
	}
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
	return cudaStatus;
}