#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <>


/**
* Matrix multiplication (CUDA Kernel) on the device: C = A * B
*/
#define TILE_WIDTH 16

__global__ void
matrixMulCUDA(float *C, float *A, float *B, int n)
{
	int k;

	// Get the row and the column in which thread resides in a block
	int row = threadIdx.x;
	int col = threadIdx.y;	
	float sum = 0.0f;
	if (row >= n || col >= n) {
		return;
	}
	for (k = 0; k < n; k++) {
		sum += A[row * n + k] * B[k * n + col];

	}
	C[row * n + col] = sum;
}

__global__ void 
matrixMulCUDA2(float *C, float *A, float *B, int n)
{
	/*
		Each thread computes more than 1 matrix elements
	*/

	// Define the starting row and ending row for each thread
	int startRow = threadIdx.y * TILE_WIDTH;
	int endRow = startRow + TILE_WIDTH;

	// Define the starting column and ending column for each thread
	int startCol = threadIdx.x * TILE_WIDTH;
	int endCol = startCol + TILE_WIDTH;

	// Now we have some block in 2 dimensions
	for (int row = startRow; row < endRow; row++) {
		for (int col = startCol; col < endCol; col++) {
			
			// Compute the proper sum for each block
			float sum = 0.0f;	// Defined as a register (Better than directly writing to C)
			for (int k = 0; k < n; k++) {
				sum += B[row * n + k] * C[k * n + col];
			}
			
			// Write back sum into C
			C[row * n + col] = sum;
		}
	}
}

void constantInit(float *data, int size, float val)
{
	for (int i = 0; i < size; ++i)
	{
		data[i] = val;
	}
}

void printMatrix(float *c, int size) {
	printf("----------------------\n");
	// Print a
	int i = 0;
	for (i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			printf("%f ", (c[i*size+j]));
		}
		printf("\n");
	}
}

/**
* Run a simple test of matrix multiplication using CUDA
*/
int matrixMultiply(int argc, char **argv, int n)
{
	// Allocate host memory for matrices A and B
	unsigned int size_A = n * n;
	unsigned int mem_size_A = sizeof(float)* size_A;
	float *h_A = (float *)malloc(mem_size_A);
	unsigned int size_B = n * n;
	unsigned int mem_size_B = sizeof(float)* size_B;
	float *h_B = (float *)malloc(mem_size_B);

	// Initialize host memory
	const float valB = 1.0f;
	constantInit(h_A, size_A, 1.0f);
	constantInit(h_B, size_B, valB);

	// Allocate device memory
	float *d_A, *d_B, *d_C;

	// Allocate host matrix C
	unsigned int mem_size_C = n * n * sizeof(float);
	float *h_C = (float *)malloc(mem_size_C);

	if (h_C == NULL)
	{
		fprintf(stderr, "Failed to allocate host matrix C!\n");
		exit(EXIT_FAILURE);
	}

	hipError_t error;

	error = hipMalloc((void **)&d_A, mem_size_A);

	if (error != hipSuccess)
	{
		printf("hipMalloc d_A returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void **)&d_B, mem_size_B);

	if (error != hipSuccess)
	{
		printf("hipMalloc d_B returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void **)&d_C, mem_size_C);

	if (error != hipSuccess)
	{
		printf("hipMalloc d_C returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	// copy host memory to device
	error = hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);

	if (error != hipSuccess)
	{
		printf("hipMemcpy (d_A,h_A) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

	if (error != hipSuccess)
	{
		printf("hipMemcpy (d_B,h_B) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	// Setup execution parameters
	dim3 gridSize(1, 1, 1);		// Number of blocks: 1
	dim3 blockSize(32, 32, 1);	// Number of threads: 1024

	// Create and start timer
	printf("Computing result using CUDA Kernel...\n");

	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	error = hipEventCreate(&start);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	hipEvent_t stop;
	error = hipEventCreate(&stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the start event
	error = hipEventRecord(start, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Execute the kernel
	matrixMulCUDA2 << < gridSize, blockSize >> > (d_C, d_A, d_B, n);

	error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to launch kernel!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the stop event
	error = hipEventRecord(stop, NULL);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Wait for the stop event to complete
	error = hipEventSynchronize(stop);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	float msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);

	printf("Elapsed time in msec = %f\n", msecTotal);

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Copy result from device to host
	error = hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

	if (error != hipSuccess)
	{
		printf("hipMemcpy (h_C,d_C) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	printMatrix(h_A, n);
	printMatrix(h_B, n);
	printMatrix(h_C, n);

	// Clean up memory
	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return EXIT_SUCCESS;

}


/**
* Program main
*/
int main(int argc, char **argv)
{
	printf("[Matrix Multiply Using CUDA] - Starting...\n");

	// By default, we use device 0
	int devID = 0;
	hipSetDevice(devID);

	hipError_t error;
	hipDeviceProp_t deviceProp;
	error = hipGetDevice(&devID);

	if (error != hipSuccess)
	{
		printf("hipGetDevice returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	}

	error = hipGetDeviceProperties(&deviceProp, devID);

	if (deviceProp.computeMode == hipComputeModeProhibited)
	{
		fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		exit(EXIT_SUCCESS);
	}

	if (error != hipSuccess)
	{
		printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
	}
	else
	{
		printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
	}

	// Size of square matrices
	size_t n = 0;
	printf("[-] N = ");
	scanf("%u", &n);

	printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", n, n, n, n);


	int matrix_result = matrixMultiply(argc, argv, n);

	system("pause");

	exit(matrix_result);
}
